#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

void fill_array(int *arr, int size){
    for(int i=0; i<size; i++){
        arr[i] = rand() % 100;
    }
}

void add_cpu(int *arr1, int *arr2, int *result, int size){
    for(int i=0; i<size; i++){
        result[i] = arr1[i] + arr2[i];
    }
}

void print_array(int *arr, int size){
    for(int i=0; i<size; i++){
        cout<<arr[i]<<" ";
    }
    cout<<endl;
}

__global__ void add(int *arr1, int *arr2, int *result, int size){
    int id = blockIdx.x;
    result[id] = arr1[id] + arr2[id];
}

int main(){
    int size;
    cout<<"Enter size of vector: ";
    cin>>size;

    int *arr1_cpu = new int[size];
    int *arr2_cpu = new int[size];
    int *result_cpu = new int[size];

    fill_array(arr1_cpu, size);
    cout<<"Array 1: ";
    print_array(arr1_cpu, size);

    fill_array(arr2_cpu, size);
    cout<<"Array 2: ";
    print_array(arr2_cpu, size);

    int *arr1_gpu, *arr2_gpu, *result_gpu;
    
    /* cudaMalloc() allocates memory from Global memory on GPU */
    hipMalloc((void **)&arr1_gpu, size*sizeof(int));
    hipMalloc((void **)&arr2_gpu, size*sizeof(int));
    hipMalloc((void **)&result_gpu, size*sizeof(int));

    /* cudaMemcpy() copies the contents from destination to source. Destination is GPU and source is CPU */
    hipMemcpy(arr1_gpu, arr1_cpu, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2_cpu, size*sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /* call to kernel. Here 'size' is number of blocks, 1 is the number of threads per block */
    add<<<size,1>>>(arr1_gpu, arr2_gpu, result_gpu, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* Here we are copying content from GPU(Device) to CPU(Host) */
    hipMemcpy(result_cpu, result_gpu, size*sizeof(int), hipMemcpyDeviceToHost);

    cout<<"\nGPU result:\n";
    print_array(result_cpu, size);
    cout<<"Elapsed Time = "<<elapsedTime<<" milliseconds"<<endl;

    hipFree(arr1_gpu);
    hipFree(arr2_gpu);
    hipFree(result_gpu);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    add_cpu(arr1_cpu, arr2_cpu, result_cpu, size);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "\nCPU result:\n";
    print_array(result_cpu, size);
    cout<<"Elapsed Time = "<<elapsedTime<<" milliseconds" << endl;

    return 0;
}