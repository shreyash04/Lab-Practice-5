#include<iostream>
#include<cstdlib>
#include<cmath>
#include<hip/hip_runtime.h>
using namespace std;

#define A_rows 2
#define A_cols 3
#define B_rows 3
#define B_cols 2
#define C_rows 2
#define C_cols 2

void matrix_multiplication_cpu(int a[A_rows][A_cols], int b[B_rows][B_cols], int c[C_rows][C_cols]){
    for(int i = 0; i < C_rows; i++){
        for(int j = 0; j < C_cols; j++){
            int sum = 0;
            for(int k = 0; k < A_cols; k++){
                sum += a[i][k] * b[k][j];
            }
            c[i][j] = sum;
        }
    }
}

__global__ void matrix_multiply(int *a, int *b, int *c)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int i;
  
    c[C_cols*y + x] = 0;
    for(i=0; i<A_cols; i++){
        c[C_cols*y + x] = c[C_cols*y + x] + a[A_cols*y + i] * b[B_cols*i + x];
    }
}


int main(){

    int A[A_rows][A_cols];
    int B[B_rows][B_cols];
    int C[C_rows][C_cols];

    for(int i = 0 ; i < A_rows; i++){
        for(int j = 0; j < A_cols; j++){
            A[i][j] = rand() % 10;
        }
    }

    for(int i = 0 ; i < B_rows; i++){
        for(int j = 0; j < B_cols; j++){
            B[i][j] = rand() % 10;
        }
    }

    int *mat1_gpu, *mat2_gpu, *result_gpu;

    cout<<"\nMatrix 1\n";
    for(int i = 0 ; i < A_rows; i++){
        for(int j = 0; j < A_cols; j++){
            cout<<A[i][j]<<" ";
        }
        cout<<endl;
    }

    cout<<"\nMatrix 2\n";
    for(int i = 0 ; i < B_rows; i++){
        for(int j = 0; j < B_cols; j++){
            cout<<B[i][j]<<" ";
        }
        cout<<endl;
    }

    hipMalloc((void **)&mat1_gpu, A_rows*A_cols*sizeof(int));
    hipMalloc((void **)&mat2_gpu, B_rows*B_cols*sizeof(int));
    hipMalloc((void **)&result_gpu, C_rows*C_cols*sizeof(int));

    hipMemcpy(mat1_gpu, A, A_rows*A_cols*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mat2_gpu, B, B_rows*B_cols*sizeof(int), hipMemcpyHostToDevice);

    /* Here we are defining two dimensional Grid(collection of blocks) structure. 
    Syntax is dim3 grid(no. of columns, no. of rows) */
    dim3 grid(B_cols, A_rows);

    float gpu_elapsed_time;
    hipEvent_t gpu_start, gpu_stop;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    matrix_multiply<<<grid, 1>>>(mat1_gpu, mat2_gpu, result_gpu);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    hipMemcpy(C, result_gpu, C_rows*C_cols*sizeof(int), hipMemcpyDeviceToHost);
    cout << "\nGPU result:\n";
    for(int i = 0 ; i < C_rows; i++){
        for(int j = 0; j < C_cols; j++){
            cout<<C[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<"GPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    matrix_multiplication_cpu(A, B, C);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    cout<<"\nCPU result:\n";
    for(int i = 0 ; i < C_rows; i++){
        for(int j = 0; j < C_cols; j++){
            cout<<C[i][j]<<" ";
        }
        cout<<endl;
    }
    cout<<"CPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    hipFree(mat1_gpu);
    hipFree(mat2_gpu);
    hipFree(result_gpu);

    return 0;
}