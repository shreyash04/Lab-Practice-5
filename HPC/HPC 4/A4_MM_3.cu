#include<iostream>
#include<cstdlib>
#include<hip/hip_runtime.h>
using namespace std;

void matrix_multiplication_cpu(int *A, int *B, int *C, int C_rows, int C_cols, int common){
    int it = 0;
    for(int row = 0; row < C_rows; row++){
        for(int col = 0; col < C_cols; col++){
            C[it] = 0;
            for(int k = 0; k < common; k++){
                C[it] += A[row*common + k] * B[C_cols*k + col];
            }
            it++;
        }
    }
}

__global__ void matrix_multiply(int *A, int *B, int *C, int C_rows, int C_cols, int common)
{
    int col = blockIdx.x;
    int row = blockIdx.y;
    int k;
  
    C[C_cols*row + col] = 0;
    for(k=0; k<common; k++){
        C[C_cols*row + col] += A[row*common + k] * B[C_cols*k + col];
    }
}


int main(){

    int A_rows, A_cols, B_rows, B_cols, C_rows, C_cols;
    cout<<"Enter A rows: ";
    cin>>A_rows;
    cout<<"Enter A cols: ";
    cin>>A_cols;
    B_rows = A_cols;
    cout<<"B rows: "<<B_rows<<endl;
    cout<<"Enter B cols: ";
    cin>>B_cols;
    C_rows = A_rows, C_cols = B_cols;
    
    int *A = new int[A_rows*A_cols];
    int *B = new int[B_rows*B_cols];
    int *C = new int[C_rows*C_cols];

    int k = 0;
    for(int i = 0 ; i < A_rows; i++){
        for(int j = 0; j < A_cols; j++){
            A[k] = rand() % 100;
            k++;
        }
    }

    k = 0;
    for(int i = 0 ; i < B_rows; i++){
        for(int j = 0; j < B_cols; j++){
            B[k] = rand() % 10;
            k++;
        }
    }

    int *mat1_gpu, *mat2_gpu, *result_gpu;

    cout<<"\nMatrix 1\n";
    k = 0;
    for(int i = 0 ; i < A_rows; i++){
        for(int j = 0; j < A_cols; j++){
            cout<<A[k]<<" ";
            k++;
        }
        cout<<endl;
    }

    cout<<"\nMatrix 2\n";
    k = 0;
    for(int i = 0 ; i < B_rows; i++){
        for(int j = 0; j < B_cols; j++){
            cout<<B[k]<<" ";
            k++;
        }
        cout<<endl;
    }

    hipMalloc((void **)&mat1_gpu, A_rows*A_cols*sizeof(int));
    hipMalloc((void **)&mat2_gpu, B_rows*B_cols*sizeof(int));
    hipMalloc((void **)&result_gpu, C_rows*C_cols*sizeof(int));

    hipMemcpy(mat1_gpu, A, A_rows*A_cols*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mat2_gpu, B, B_rows*B_cols*sizeof(int), hipMemcpyHostToDevice);

    /* Here we are defining two dimensional Grid(collection of blocks) structure. 
    Syntax is dim3 grid(no. of columns, no. of rows) */
    dim3 grid(B_cols, A_rows);

    float gpu_elapsed_time;
    hipEvent_t gpu_start, gpu_stop;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    matrix_multiply<<<grid, 1>>>(mat1_gpu, mat2_gpu, result_gpu, C_rows, C_cols, A_cols);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    hipMemcpy(C, result_gpu, C_rows*C_cols*sizeof(int), hipMemcpyDeviceToHost);
    cout << "\nGPU result:\n";
    k = 0;
    for(int i = 0 ; i < C_rows; i++){
        for(int j = 0; j < C_cols; j++){
            cout<<C[k]<<" ";
            k++;
        }
        cout<<endl;
    }
    cout<<"GPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    matrix_multiplication_cpu(A, B, C, C_rows, C_cols, A_cols);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    cout<<"\nCPU result:\n";
    k = 0;
    for(int i = 0 ; i < C_rows; i++){
        for(int j = 0; j < C_cols; j++){
            cout<<C[k]<<" ";
            k++;
        }
        cout<<endl;
    }
    cout<<"CPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    hipFree(mat1_gpu);
    hipFree(mat2_gpu);
    hipFree(result_gpu);

    return 0;
}